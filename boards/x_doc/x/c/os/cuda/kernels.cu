#include "hip/hip_runtime.h"
#include "kernels.h"

/****************************************** Partie GPU ***********************************************/



//seulement pour tailles multiples de BLOCK_SZ
__global__ void KVectAddFast(const float *A, const float*B, float*C)
{
size_t idx=threadIdx.x+blockIdx.x*blockDim.x;
C[idx]=A[idx]+B[idx];

}

//toutes les tailles
__global__ void KVectAdd(const float *A, const float*B, float*C, size_t vsize)
{
size_t idx=threadIdx.x+blockIdx.x*blockDim.x;
if (idx<vsize) C[idx]=A[idx]+B[idx];

}

__global__ void KImgInv(const float *A, float*B)
{
	size_t idx=blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	B[idx]=1.0-A[idx];
}

/****************************************** Partie CPU ********************************************/
bool LaunchVectAdd(const float*d_A, const float*d_B, float* d_C, size_t vsize)
{
  const int BLOCK_SZ=256;
  if (vsize%BLOCK_SZ==0)
  {
    dim3 Db(BLOCK_SZ,1,1), Dg(vsize/BLOCK_SZ,1,1);
    KVectAddFast<<<Dg,Db>>>(d_A, d_B, d_C);
  }
  else
  {
    dim3 Db(BLOCK_SZ,1,1), Dg((vsize+BLOCK_SZ-1)/BLOCK_SZ,1,1);
    KVectAdd<<<Dg,Db>>>(d_A, d_B, d_C,vsize);
  }

    return TestLastError ("Lauch kernel VectAdd");
}

bool LaunchImgInv (const float*d_A,float*d_B, size_t vsize)
{
	const int BLOCK_SZ_x=128,BLOCK_SZ_y=4;
	dim3 Db(BLOCK_SZ_x,BLOCK_SZ_y,1), Dg(vsize/(BLOCK_SZ_x*BLOCK_SZ_y),1,1);
	KImgInv<<<Dg,Db>>>(d_A,d_B);
	
	return TestLastError ("Lauch Kernel ImgInv");
}
